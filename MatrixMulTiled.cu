
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define M 1024   // Number of rows in matrix A and resulting matrix C
#define K 1024   // Number of columns in matrix A and rows in matrix B
#define N 1024   // Number of columns in matrix B

#define TILE_SIZE 32   // Tile size for tiling

__global__ void matrixMulTiled(float* A, float* B, float* C)
{
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float sum = 0.0f;

    for (int t = 0; t < (K - 1) / TILE_SIZE + 1; t++) {
        if (row < M && t * TILE_SIZE + tx < K) {
            As[ty][tx] = A[row * K + t * TILE_SIZE + tx];
        } else {
            As[ty][tx] = 0.0f;
        }

        if (t * TILE_SIZE + ty < K && col < N) {
            Bs[ty][tx] = B[(t * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++) {
            sum += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

int main()
{
    float *h_A, *h_B, *h_C;  // Host matrices
    float *d_A, *d_B, *d_C;  // Device matrices

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    // Allocate memory for host matrices
    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C = (float*)malloc(size_C);

    // Initialize host matrices
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            h_A[i * K + j] = i + j;
        }
    }
    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            h_B[i * N + j] = i - j;
        }
    }

    // Allocate memory for device matrices
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Copy host matrices to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((N - 1) / TILE_SIZE + 1, (M - 1) / TILE_SIZE + 1);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start, 0);

    // Launch kernel for matrix multiplication using tiling
    matrixMulTiled<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Record end time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time in milliseconds
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Matrix multiplication using tiling successful! Elapsed time: %.3f ms\n", elapsedTime);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += h_A[i * K + k] * h_B[k * N + j];
            }
            if (fabs(h_C[i * N + j] - sum) > 1e-5) {
                printf("Error: mismatch at (%d, %d): expected %f, actual %f\n",
                       i, j, sum, h_C[i * N + j]);
                return -1;
            }
        }
    }

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
